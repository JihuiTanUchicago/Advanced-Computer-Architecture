#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <algorithm>
#include <chrono>

#define DATA_SIZE (1 << 24)
#define NUM_BINS 256

__global__ void histogram_coarse_gpu(const int* data, int* bins, int size, int chunksize) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    int start = tid * chunksize;
    int end = min(start + chunksize, size);
    for (int i = start; i < end; ++i) {
        atomicAdd(&bins[data[i]], 1);
    }
}

void generate_data(int* data) {
    srand(42);
    for (int i = 0; i < DATA_SIZE; ++i) {
        data[i] = rand() % NUM_BINS;
    }
}

int main() {
    int* data_h = (int*)malloc(DATA_SIZE * sizeof(int));
    generate_data(data_h);

    int* data_d; hipMalloc(&data_d, DATA_SIZE * sizeof(int));
    int* bins_d; hipMalloc(&bins_d, NUM_BINS * sizeof(int));
    hipMemcpy(data_d, data_h, DATA_SIZE * sizeof(int), hipMemcpyHostToDevice);
    hipMemset(bins_d, 0, NUM_BINS * sizeof(int));

    int blockSize = 256;
    int chunksize = 64 * blockSize;
    int numBlocks = (DATA_SIZE + chunksize - 1) / chunksize;

    auto start = std::chrono::high_resolution_clock::now();
    histogram_coarse_gpu<<<numBlocks, blockSize>>>(data_d, bins_d, DATA_SIZE, chunksize);
    hipDeviceSynchronize();
    auto end = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double> elapsed = end - start;
    printf("GPU Coarse Runtime: %.6f seconds\n", elapsed.count());

    hipFree(data_d);
    hipFree(bins_d);
    free(data_h);
    return 0;
}
