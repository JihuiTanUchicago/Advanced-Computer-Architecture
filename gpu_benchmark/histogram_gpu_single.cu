#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <algorithm>
#include <chrono>

#define DATA_SIZE (1 << 24)
#define NUM_BINS 256

__global__ void histogram_single_thread_gpu(const int* data, int* bins, int size) {
    for (int i = 0; i < size; ++i) {
        atomicAdd(&bins[data[i]], 1);
    }
}

void generate_data(int* data) {
    srand(42);
    for (int i = 0; i < DATA_SIZE; ++i) {
        data[i] = rand() % NUM_BINS;
    }
}

int main() {
    int* data_h = (int*)malloc(DATA_SIZE * sizeof(int));
    generate_data(data_h);

    int* data_d; hipMalloc(&data_d, DATA_SIZE * sizeof(int));
    int* bins_d; hipMalloc(&bins_d, NUM_BINS * sizeof(int));
    hipMemcpy(data_d, data_h, DATA_SIZE * sizeof(int), hipMemcpyHostToDevice);
    hipMemset(bins_d, 0, NUM_BINS * sizeof(int));

    auto start = std::chrono::high_resolution_clock::now();
    histogram_single_thread_gpu<<<1, 1>>>(data_d, bins_d, DATA_SIZE);
    hipDeviceSynchronize();
    auto end = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double> elapsed = end - start;
    printf("GPU Single Thread Runtime: %.6f seconds\n", elapsed.count());

    hipFree(data_d);
    hipFree(bins_d);
    free(data_h);
    return 0;
}
