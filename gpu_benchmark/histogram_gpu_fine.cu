#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <algorithm>
#include <chrono>

#define DATA_SIZE (1 << 24)
#define NUM_BINS 256

__global__ void histogram_fine_gpu(const int* data, int* bins, int size) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    if (i < size) {
        atomicAdd(&bins[data[i]], 1);
    }
}

void generate_data(int* data) {
    srand(42);
    for (int i = 0; i < DATA_SIZE; ++i) {
        data[i] = rand() % NUM_BINS;
    }
}

int main() {
    int* data_h = (int*)malloc(DATA_SIZE * sizeof(int));
    generate_data(data_h);

    int* data_d; hipMalloc(&data_d, DATA_SIZE * sizeof(int));
    int* bins_d; hipMalloc(&bins_d, NUM_BINS * sizeof(int));
    hipMemcpy(data_d, data_h, DATA_SIZE * sizeof(int), hipMemcpyHostToDevice);
    hipMemset(bins_d, 0, NUM_BINS * sizeof(int));

    int blockSize = 1024;
    int numBlocks = (DATA_SIZE + blockSize - 1) / blockSize;

    auto start = std::chrono::high_resolution_clock::now();
    histogram_fine_gpu<<<numBlocks, blockSize>>>(data_d, bins_d, DATA_SIZE);
    hipDeviceSynchronize();
    auto end = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double> elapsed = end - start;
    printf("GPU Fine Runtime: %.6f seconds\n", elapsed.count());

    hipFree(data_d);
    hipFree(bins_d);
    free(data_h);
    return 0;
}
